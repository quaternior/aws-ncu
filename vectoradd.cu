// file: vector_add.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <random>
#include <algorithm>
#include <cmath>

#define CUDA_CHECK(call)                                                        \
  do {                                                                          \
    hipError_t err__ = (call);                                                 \
    if (err__ != hipSuccess) {                                                 \
      std::fprintf(stderr, "CUDA error %s at %s:%d\n",                          \
                   hipGetErrorString(err__), __FILE__, __LINE__);              \
      std::exit(EXIT_FAILURE);                                                  \
    }                                                                           \
  } while (0)

// Grid-stride kernel: y[i] = a[i] + b[i]
__global__ void vecAdd_kernel(const float* __restrict__ a,
                              const float* __restrict__ b,
                              float* __restrict__ y,
                              size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * (size_t)gridDim.x;
  for (size_t i = idx; i < n; i += stride) {
    y[i] = a[i] + b[i];
  }
}

int main(int argc, char** argv) {
  // problem size (default 1<<24)
  size_t N = (argc > 1) ? static_cast<size_t>(atoll(argv[1])) : (1ull << 24);
  std::printf("N = %zu\n", N);

  // host init
  std::vector<float> hA(N), hB(N), hY(N), hRef(N);
  std::mt19937 rng(123);
  std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
  for (size_t i = 0; i < N; ++i) {
    hA[i] = dist(rng);
    hB[i] = dist(rng);
    hRef[i] = hA[i] + hB[i];
  }

  // device alloc
  float *dA = nullptr, *dB = nullptr, *dY = nullptr;
  CUDA_CHECK(hipMalloc(&dA, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&dB, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&dY, N * sizeof(float)));

  // H2D
  CUDA_CHECK(hipMemcpy(dA, hA.data(), N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dB, hB.data(), N * sizeof(float), hipMemcpyHostToDevice));

  // launch
  int block = 256;
  int maxBlocks;
  CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxBlocks, vecAdd_kernel, block, 0));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
  int grid = std::min<int>(prop.multiProcessorCount * maxBlocks, (int)((N + block - 1) / block));
  grid = std::max(grid, 1);  // ensure at least 1 block

  vecAdd_kernel<<<grid, block>>>(dA, dB, dY, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // D2H
  CUDA_CHECK(hipMemcpy(hY.data(), dY, N * sizeof(float), hipMemcpyDeviceToHost));

  // check
  double max_abs_err = 0.0;
  for (size_t i = 0; i < N; ++i) {
    max_abs_err = std::max(max_abs_err, (double)std::abs(hY[i] - hRef[i]));
  }
  std::printf("Max |error| = %.3e\n", max_abs_err);

  // cleanup
  CUDA_CHECK(hipFree(dA));
  CUDA_CHECK(hipFree(dB));
  CUDA_CHECK(hipFree(dY));

  if (max_abs_err > 1e-6) {
    std::fprintf(stderr, "Validation failed.\n");
    return EXIT_FAILURE;
  }
  std::puts("OK");
  return EXIT_SUCCESS;
}
